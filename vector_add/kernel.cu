#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <SDL2/SDL.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 64 // Size of the grid (NxN)
#define CELL_SIZE 10 // Size of each cell in pixels
#define WINDOW_SIZE (N * CELL_SIZE)
#define NUM_GENES 64

struct Cell {
    int alive; // Alive/Dead
    int energy; // Energy level
    int age; // Age
    int genes[NUM_GENES]; // Array of genes
    int currentActivity; // Current activity
};

__device__ int getCellAlive(Cell* grid, int x, int y) {
    return grid[y * N + x].alive;
}

__device__ void setCellAlive(Cell* grid, int x, int y, int value) {
    grid[y * N + x].alive = value;
}

__global__ void gameOfLifeKernel(Cell* currentGrid, Cell* nextGrid) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < N && y < N) {
        int liveNeighbors = 0;
        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                if (i == 0 && j == 0) continue;
                int neighborX = (x + i + N) % N;
                int neighborY = (y + j + N) % N;
                liveNeighbors += getCellAlive(currentGrid, neighborX, neighborY);
            }
        }

        Cell currentState = currentGrid[y * N + x];
        Cell nextState = currentState;
        if (currentState.alive == 1) {
            if (currentState.age > 100) nextState.alive = 0;
            if (liveNeighbors < 2 || liveNeighbors > 3) nextState.alive = 0;
            nextState.age++;
            nextState.energy--; // Example: reduce energy over time
        }
        else {
            if (liveNeighbors == 3) nextState.alive = 1;
            nextState.age = 0;
            nextState.energy = 100; // Example: reset energy on birth
        }
        nextGrid[y * N + x] = nextState;
    }
}

void renderGrid(SDL_Renderer* renderer, Cell* grid) {
    SDL_SetRenderDrawColor(renderer, 0, 0, 0, 255);
    SDL_RenderClear(renderer);

    SDL_SetRenderDrawColor(renderer, 0, 255, 0, 255);
    for (int y = 0; y < N; ++y) {
        for (int x = 0; x < N; ++x) {
            if (grid[y * N + x].alive) {
                SDL_Rect cell;
                cell.x = x * CELL_SIZE;
                cell.y = y * CELL_SIZE;
                cell.w = CELL_SIZE;
                cell.h = CELL_SIZE;
                SDL_RenderFillRect(renderer, &cell);
            }
        }
    }

    SDL_RenderPresent(renderer);
}

int main(int argc, char* argv[]) {
    Cell* currentGrid;
    Cell* nextGrid;
    hipMallocManaged(&currentGrid, N * N * sizeof(Cell));
    hipMallocManaged(&nextGrid, N * N * sizeof(Cell));

    // Initialize the grid with random pattern
    srand(time(NULL));
    for (int y = 0; y < N; ++y) {
        for (int x = 0; x < N; ++x) {
            currentGrid[y * N + x].alive = rand() % 2;
            currentGrid[y * N + x].energy = rand() % 100;
            currentGrid[y * N + x].age = rand() % 100;
            currentGrid[y * N + x].currentActivity = rand() % 10;
            for (int i = 0; i < NUM_GENES; ++i) {
                currentGrid[y * N + x].genes[i] = rand() % 2;
            }
        }
    }

    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        fprintf(stderr, "Could not initialize SDL: %s\n", SDL_GetError());
        return 1;
    }

    SDL_Window* window = SDL_CreateWindow("Game of Life",
        SDL_WINDOWPOS_UNDEFINED,
        SDL_WINDOWPOS_UNDEFINED,
        WINDOW_SIZE,
        WINDOW_SIZE,
        SDL_WINDOW_SHOWN);
    if (!window) {
        fprintf(stderr, "Could not create window: %s\n", SDL_GetError());
        SDL_Quit();
        return 1;
    }

    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    if (!renderer) {
        fprintf(stderr, "Could not create renderer: %s\n", SDL_GetError());
        SDL_DestroyWindow(window);
        SDL_Quit();
        return 1;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    bool quit = false;
    SDL_Event event;

    while (!quit) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                quit = true;
            }
        }

        gameOfLifeKernel << <numBlocks, threadsPerBlock >> > (currentGrid, nextGrid);
        hipDeviceSynchronize();

        Cell* temp = currentGrid;
        currentGrid = nextGrid;
        nextGrid = temp;

        renderGrid(renderer, currentGrid);
        //SDL_Delay(100); // Delay for 100ms
    }

    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    hipFree(currentGrid);
    hipFree(nextGrid);

    return 0;
}
